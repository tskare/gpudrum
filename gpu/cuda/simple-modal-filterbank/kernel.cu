#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_complex.h"

#include <stdio.h>

#include <windows.h>
#include <tchar.h>

// Synchronization primitives for communicating with plugin client.
// (or any other process that uses these named objects).
TCHAR szName[] = TEXT("Local\\GPUModalBankMem");
TCHAR szNameSemaphore[] = TEXT("Local\\GPUModalBankSemaphore");
TCHAR szNameSemaphoreGPU[] = TEXT("Local\\GPUModalBankSemaphoreGPU");
#define SHAREDMEMSIZE 1024*512*2

#define BUFFERSIZE 256
constexpr int NDRUMS = 10;
constexpr int NMODES = NDRUMS * 1024;
constexpr int NWARPS = 320;  // 10240/32

static float host_samplebuffer[BUFFERSIZE*NWARPS*2];

// Shared Memory Layout
// ----- First Section: Input parameters -----
// 10000 modes *
struct ModeInfo {
	bool enabled;
	bool reset;

	bool amp_changed;
	float amp_real;
	float amp_imag;
	
	float damp;
	float freq;
	bool freq_changed;
};  // approximately 8 * 4 bytes = 32 bytes
// all modes total = 320000 = 320KB

// ----- Drum info ----
// Cleanup: should reorder these sections, but these were added chronologically.
// 10 drums * 8 controllable params per drum.

/// ----- Input State -----
// NDRUMS times BUFFERSIZE for inputs. = 10240 = 10KB

// ----- Third Section: Audio output to Host -----
// 4 bytes per sample * 1024 buffer size (supports stereo@512) = 4K


__device__ __forceinline__ hipFloatComplex custom_cexpf(hipFloatComplex z) {
	hipComplex res;
	float t = expf(z.x);
	sincosf(z.y, &res.y, &res.x);
	res.x *= t;
	res.y *= t;
	return res;
}

__global__ void filterbankKernel(float *yprev, const ModeInfo *mi, const float* drumInfo, const float* input, float* output) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int whichwarp = (int)(i / 32);
	bool is_first_thread_in_warp = (i % 32) == 0;

	// Init - pull from shared memory.
	hipComplex y;
	y.x = yprev[2 * i];
	y.y = yprev[2 * i + 1];

	if (mi[i].reset) {
		y.x = 0.0f;
		y.y = 0.0f;
	}

	hipComplex input_amp;
	input_amp.x = mi[i].amp_real;
	input_amp.y = mi[i].amp_real;

	hipComplex input_complex;

	hipComplex exp_term;
	{
		// regenerate
		hipComplex e_stuff;
		e_stuff.x = -mi[i].damp;
		e_stuff.y = mi[i].freq;
		exp_term = custom_cexpf(e_stuff);
	}

	int whichDrum = (int)(i / 1024);
	float pan = drumInfo[whichDrum * 8 + 0];

	const float *input_base = input + (BUFFERSIZE*whichDrum);
	// Main loop - spin for enough cycles to generate the whole buffer.
	for (int samp = 0; samp < BUFFERSIZE; samp++) {
		y = hipCmulf(exp_term, y);
		// Always assume input is present -- host will set amplitude to 0 when not.
		// This could be optimized with an "input present" flag given it's usually 0.
		input_complex.x = input_base[samp];
		input_complex.y = 0.0f;
		y = hipCaddf(y, hipCmulf(input_complex, input_amp));

		// Tree-sum, channels interleaved
		float merge_output_L = y.x*pan;
		float merge_output_R = y.x*(1-pan);
		for (int offset = 16; offset > 0; offset /= 2) {
			merge_output_L += __shfl_down_sync(0xffffffff, merge_output_L, offset);
			merge_output_R += __shfl_down_sync(0xffffffff, merge_output_R, offset);
		}
		if (is_first_thread_in_warp) {
			output[whichwarp * (BUFFERSIZE*2) + 2*samp] = merge_output_L;
			output[whichwarp * (BUFFERSIZE * 2) + 2*samp + 1] = merge_output_R;
		}

		// Mono demos:
		/*
		float merge_output = y.x;
		for (int offset = 16; offset > 0; offset /= 2) {
			merge_output += __shfl_down_sync(0xffffffff, merge_output, offset);
		}
		if (is_first_thread_in_warp) {
			output[whichwarp * BUFFERSIZE + 2*samp] = merge_output;
		}
		*/
	}

	// Save state back to shared/global memory for next kernel invocation.
	yprev[2 * i] = y.x;
	yprev[2 * i + 1] = y.y;
}

int main()
{
	HANDLE hMapFile = CreateFileMapping(
		INVALID_HANDLE_VALUE, // use paging file,
		NULL, // default security
		PAGE_READWRITE,
		0, // max objeect size (high-order)
		SHAREDMEMSIZE,  // max obj size (low-order),
		szName);
	if (hMapFile == nullptr) {
		fprintf(stderr, "shared memory init failed! %d", GetLastError());
		return 1;
	}
	LPCTSTR pBuf = (LPTSTR)MapViewOfFile(hMapFile,
		FILE_MAP_ALL_ACCESS,
		0,
		0,
		SHAREDMEMSIZE);
	if (pBuf == nullptr) {
		fprintf(stderr, "mapviewoffile failed! %d", GetLastError());
		CloseHandle(hMapFile);
		return 1;
	}

	HANDLE hSemaphore = CreateSemaphoreA(NULL, 0, 1, szNameSemaphore);
	if (hSemaphore == nullptr) {
		fprintf(stderr, "could not create semaphore %d", GetLastError());
		CloseHandle(hMapFile);
		return 1;
	}
	HANDLE hSemaphoreGPU = CreateSemaphoreA(NULL, 0, 1, szNameSemaphoreGPU);
	if (hSemaphore == nullptr) {
		fprintf(stderr, "could not create semaphore-gpu %d", GetLastError());
		CloseHandle(hMapFile);
		return 1;
	}

	// Init all our buffers
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	float* dev_previousvalues; // previous values of exponential across kernel launches. Interleaved complex.
	int* dev_modeinfo;  // modeinfo, per-drum.
	float* dev_druminfo;  // drum info, per-drum
	float* dev_inputs;  // input signals, per-drum
	float* dev_output_samps;  // output samples, per-warp
	
	cudaStatus = hipMalloc((void**)&dev_previousvalues, NMODES * 2 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_previousvalues failed!");
		return 1;
	}
	cudaStatus = hipMalloc((void**)&dev_modeinfo, NMODES * sizeof(ModeInfo));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_modeinfo failed!");
		return 1;
	}
	// 8 params per drum
	cudaStatus = hipMalloc((void**)&dev_druminfo, NDRUMS * sizeof(float) * 8);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_druminfo failed!");
		return 1;
	}
	cudaStatus = hipMalloc((void**)&dev_inputs, NDRUMS * BUFFERSIZE * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_inputs failed!");
		return 1;
	}
	cudaStatus = hipMalloc((void**)&dev_output_samps, NWARPS * 2 * BUFFERSIZE * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc output_samps failed!");
		return 1;
	}

	int times = 0;
	
	int* sharedmem_modeinfoptr = (int*)pBuf;
	int* sharedmem_druminfoptr = (int*)((char*)sharedmem_modeinfoptr + NMODES * sizeof(ModeInfo));
	int* sharedmem_inputptr = (int*)((char*)sharedmem_druminfoptr + NDRUMS*8*sizeof(float));
	int* sharedmem_outputptr = (int*)((char*)sharedmem_inputptr + NDRUMS * BUFFERSIZE * sizeof(float));
	fprintf(stderr, "gpuaudio kernel process: starting main loop. Ctrl-C to exit.\n");
	while (true) {
		times++;
		WaitForSingleObject(hSemaphore, INFINITE);

		// Copy modes from shared memory to device.
		cudaStatus = hipMemcpy(dev_modeinfo, sharedmem_modeinfoptr, NMODES * sizeof(ModeInfo), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy modeinfos failed!");
			return 1;
		}
		cudaStatus = hipMemcpy(dev_druminfo, sharedmem_druminfoptr, NDRUMS * 8*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy drumInfos failed!");
			return 1;
		}
		cudaStatus = hipMemcpy(dev_inputs, sharedmem_inputptr, NDRUMS * BUFFERSIZE * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy inputs failed!");
			return 1;
		}

		// Kernel launch
		// NMODES total. (10 drums * 1024)
		filterbankKernel << <10, 1024>> > (dev_previousvalues, (ModeInfo*)dev_modeinfo, dev_druminfo, dev_inputs, dev_output_samps);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return 1;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
			return 1;
		}
		
		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(host_samplebuffer, dev_output_samps, BUFFERSIZE*NWARPS*2*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy samples-back failed!");
			return 1;
		}

		// Sum up and output to buffer
		float* sampsBuf = ((float*)sharedmem_outputptr);
		for (int samplei = 0; samplei < BUFFERSIZE; samplei++) {
			float sampleL = 0.0f;
			float sampleR = 0.0f;
			for (int j = 0; j < NWARPS; j++) {
				sampleL += host_samplebuffer[j*(BUFFERSIZE*2) + 2*samplei + 0];
				sampleR += host_samplebuffer[j *(BUFFERSIZE*2) + 2 * samplei + 1];
			}
			sampsBuf[2*samplei+0] = sampleL;
			sampsBuf[2*samplei+1] = sampleR;
		}
		ReleaseSemaphore(hSemaphoreGPU, 1, NULL);
	}
    return 0;
}
